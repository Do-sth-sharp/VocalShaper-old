#include "hip/hip_runtime.h"
﻿#include "KarPlusStrong.cuh"
#include <>

#if CUDA_ENABLED

__host__ hipError_t doSynthesis(
	const float* part, int partSize,
	float* buffer, int bufferSize,
	const int* unitArray, int unitSize)
{
	//使用设备0
	int device = 0;
	if (auto result = hipSetDevice(device)) {
		return result;
	}

	//获取设备属性
	hipDeviceProp_t deviceProp;
	if (auto result = hipGetDeviceProperties(&deviceProp, device)) {
		return result;
	}

	//在设备上申请合成结果储存空间
	float* deviceBufferPtr = nullptr;
	if (auto result = hipMalloc(&deviceBufferPtr, bufferSize * sizeof(float))) {
		return result;
	}

	//在设备上申请原始合成单元储存空间
	float* devicePartPtr = nullptr;
	if (auto result = hipMalloc(&devicePartPtr, partSize * sizeof(float))) {
		hipFree(deviceBufferPtr);
		return result;
	}

	//在设备上申请合成单元滤波后序列的储存空间
	float* deviceUnitPtr = nullptr;
	if (auto result = hipMalloc(&deviceUnitPtr, (size_t)unitSize * partSize * sizeof(float))) {
		hipFree(deviceBufferPtr);
		hipFree(devicePartPtr);
		return result;
	}

	//将原始合成单元存入设备
	if (auto result = hipMemcpy(devicePartPtr, part, partSize * sizeof(float), hipMemcpyHostToDevice)) {
		hipFree(deviceBufferPtr);
		hipFree(devicePartPtr);
		hipFree(deviceUnitPtr);
		return result;
	}

	//对合成单元滤波生成目标单元
	if (auto result = computeEachUnit(deviceProp, deviceUnitPtr, devicePartPtr, unitSize, partSize)) {
		hipFree(deviceBufferPtr);
		hipFree(devicePartPtr);
		hipFree(deviceUnitPtr);
		return result;
	}

	//释放原始合成单元
	hipFree(devicePartPtr);

	//TODO 单元拼接
	
	//返回成功
	return hipError_t::hipSuccess;
}

__host__ hipError_t computeEachUnit(
	const hipDeviceProp_t& prop,
	float* unitMem, const float* unit,
	int count, int unitLength)
{
	//无需计算
	if (unitLength == 0) { return hipError_t::hipErrorUnknown; }

	//获取设备属性
	auto& blockSizeInGrid = prop.maxGridSize;
	auto& threadSizeInBlock = prop.maxThreadsDim;
	int blockNumInAGrid = blockSizeInGrid[0] * blockSizeInGrid[1] * blockSizeInGrid[2];
	int threadNumInABlock = threadSizeInBlock[0] * threadSizeInBlock[1] * threadSizeInBlock[2];

	//计算block, thread用量与循环轮数
	int xSize = blockSizeInGrid[0], ySize = blockSizeInGrid[1], zSize = blockSizeInGrid[2];
	int txSize = threadSizeInBlock[0], tySize = threadSizeInBlock[1], tzSize = threadSizeInBlock[2];
	int roundSize = (unitLength / threadNumInABlock) / blockNumInAGrid + 1;
	if (roundSize == 1) {
		zSize = (unitLength / threadNumInABlock) / (blockSizeInGrid[0] * blockSizeInGrid[1]) + 1;
		if (zSize == 1) {
			ySize = (unitLength / threadNumInABlock) / blockSizeInGrid[0] + 1;
			if (ySize == 1) {
				xSize = (unitLength / threadNumInABlock) + 1;
				if (xSize == 1) {
					tzSize = unitLength / (threadSizeInBlock[0] * threadSizeInBlock[1]) + 1;
					if (tzSize == 1) {
						tySize = unitLength / threadSizeInBlock[0] + 1;
						if (tySize == 1) {
							txSize = unitLength;
						}
					}
				}
			}
		}
	}

	//规划算子
	dim3 grid(xSize, ySize, zSize), block(txSize, tySize, tzSize);

	//循环计算每一个单元
	for (int i = 0; i < count; i++) {
		if (i == 0) {
			for (int r = 0; r < roundSize; r++) {
				computeUnit <<<grid, block>>> (
					&unitMem[i * unitLength + r * blockNumInAGrid * threadNumInABlock],
					&unit[r * blockNumInAGrid * threadNumInABlock],
					unitLength);
			}
		}
		else {
			for (int r = 0; r < roundSize; r++) {
				computeUnit <<<grid, block>>> (
					&unitMem[i * unitLength + r * blockNumInAGrid * threadNumInABlock],
					&unitMem[(i - 1) * unitLength + r * blockNumInAGrid * threadNumInABlock],
					unitLength);
			}
		}
		hipDeviceSynchronize();
	}

	return hipError_t::hipSuccess;
}

__global__ void computeUnit(
	float* unitMemBase, const float* unitBase, int unitLength)
{
	//定位GPU线程
	int blockIndex = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int blockSize = blockDim.x * blockDim.y * blockDim.z;
	int threadIndex = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * blockDim.x + threadIdx.x;
	int index = blockIndex * blockSize + threadIndex;

	//定位处理目标
	if (index >= unitLength || index < 0) { return; }
	int nextIndex = index + 1;
	if (index == unitLength - 1) { nextIndex = 0; }

	//平均
	unitMemBase[index] = unitBase[index] + (unitBase[nextIndex] - unitBase[index]) / 2;
}

#endif