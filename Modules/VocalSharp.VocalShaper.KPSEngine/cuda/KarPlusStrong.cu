#include "hip/hip_runtime.h"
﻿#include "KarPlusStrong.cuh"
#include <>

#if CUDA_ENABLED

//滤波单位算子
__global__ void computeUnit(
	float* unitMemBase, const float* unitBase, int unitLength, int totalThread);

//衰减单位算子
__global__ void attenuateUnit(
	float* unitMemBase, int unitLength, float sCoefficient, float eCoefficient, int totalThread);

__host__ hipError_t doSynthesisGPU(
	const float* part, int partSize,
	float* buffer, int bufferSize,
	const int* unitArray, int unitSize)
{
	//无需计算
	if (partSize <= 1) { return hipError_t::hipErrorUnknown; }
	if (bufferSize <= 1) { return hipError_t::hipErrorUnknown; }
	if (unitSize == 0) { return hipError_t::hipErrorUnknown; }

	//计算合成单元最多用到的长度
	int maxUnitLength = 0;
	for (int i = 0; i < unitSize; i++) {
		if (unitArray[i] > maxUnitLength) { maxUnitLength = unitArray[i]; }
	}
	if (maxUnitLength > partSize) { return hipError_t::hipErrorUnknown; }

	//选择设备
	int device = -1;
	hipDeviceProp_t deviceProp = cudaDevicePropDontCare;
	if (auto result = hipChooseDevice(&device, &deviceProp)) {
		return result;
	}
	if (auto result = hipGetDeviceProperties(&deviceProp, device)) {
		return result;
	}
	if (auto result = hipSetDevice(device)) {
		return result;
	}

	//在主机上申请合成结果储存空间
	float* hostBufferPtr = nullptr;
	if (auto result = hipHostMalloc(&hostBufferPtr, bufferSize * sizeof(float), hipMemAttachGlobal)) {
		hipDeviceReset();
		return result;
	}

	//在设备上申请原始合成单元储存空间
	float* devicePartPtr = nullptr;
	if (auto result = hipMalloc(&devicePartPtr, maxUnitLength * sizeof(float))) {
		hipHostFree(hostBufferPtr);
		hipDeviceReset();
		return result;
	}

	//在设备上申请合成单元滤波后序列的储存空间
	float* deviceUnitPtr = nullptr;
	if (auto result = hipMalloc(&deviceUnitPtr, (size_t)unitSize * maxUnitLength * sizeof(float))) {
		hipHostFree(hostBufferPtr);
		hipFree(devicePartPtr);
		hipDeviceReset();
		return result;
	}

	//将原始合成单元存入设备
	if (auto result = hipMemcpy(devicePartPtr, part, maxUnitLength * sizeof(float), hipMemcpyHostToDevice)) {
		hipHostFree(hostBufferPtr);
		hipFree(devicePartPtr);
		hipFree(deviceUnitPtr);
		hipDeviceReset();
		return result;
	}

	//建立流
	hipStream_t* streams = new hipStream_t[unitSize];
	if (!streams) {
		hipHostFree(hostBufferPtr);
		hipFree(devicePartPtr);
		hipFree(deviceUnitPtr);
		hipDeviceReset();
		return hipError_t::hipErrorOutOfMemory;
	}
	for (int i = 0; i < unitSize; i++) {
		hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
	}

	//建立同步事件
	hipEvent_t* events = new hipEvent_t[unitSize];
	if (!events) {
		//销毁流
		for (int i = 0; i < unitSize; i++) {
			hipStreamDestroy(streams[i]);
		}
		hipHostFree(hostBufferPtr);
		hipFree(devicePartPtr);
		hipFree(deviceUnitPtr);
		delete[] streams;
		hipDeviceReset();
		return hipError_t::hipErrorOutOfMemory;
	}
	for (int i = 0; i < unitSize; i++) {
		hipEventCreateWithFlags(&events[i], hipEventDisableTiming);
	}

	//等待同步
	hipDeviceSynchronize();

	//获取设备属性
	int threadNumInABlock = deviceProp.maxThreadsPerBlock;

	//计算循环轮数
	int roundSize = (maxUnitLength / threadNumInABlock) + 1;

	//深度优先建立流计算任务
	int outDeviation = 0;
	for (int i = 0; i < unitSize; i++) {
		//在开始任务一前确认上一流已到达事件同步点
		if (i > 0) {
			hipStreamWaitEvent(streams[i], events[i - 1]);
		}

		//任务一：对拼接单元依次低通滤波
		if (i == 0) {
			computeUnit <<<roundSize, threadNumInABlock, 0, streams[i]>>> (
				&deviceUnitPtr[i * maxUnitLength],
				&devicePartPtr[0], maxUnitLength, threadNumInABlock);
		}
		else {
			computeUnit <<<roundSize, threadNumInABlock, 0, streams[i]>>> (
				&deviceUnitPtr[i * maxUnitLength],
				&deviceUnitPtr[(i - 1) * maxUnitLength], maxUnitLength, threadNumInABlock);
		}

		//设置事件同步点
		hipEventRecord(events[i], streams[i]);

		//在开始任务二前确认下一流已到达事件同步点
		if (i < unitSize - 1) {
			hipStreamWaitEvent(streams[i], events[i + 1]);
		}

		//计算拼接单元长度
		int currentUnitLength = unitArray[i];

		//任务二：线性衰减
		int partRoundSize = (currentUnitLength / threadNumInABlock) + 1;
		attenuateUnit <<<partRoundSize, threadNumInABlock, 0, streams[i]>>> (
			&deviceUnitPtr[i * maxUnitLength], currentUnitLength,
			1.f - outDeviation / (float)(bufferSize - 1),
			1.f - (outDeviation + currentUnitLength - 1) / (float)(bufferSize - 1), threadNumInABlock);

		//任务三：将拼接单元拷贝到指定位置
		hipMemcpyAsync(
			&hostBufferPtr[outDeviation], &deviceUnitPtr[i * maxUnitLength], currentUnitLength, hipMemcpyDeviceToHost, streams[i]);
		outDeviation += currentUnitLength;
	}

	//等待同步
	hipDeviceSynchronize();

	//销毁事件
	for (int i = 0; i < unitSize; i++) {
		hipEventDestroy(events[i]);
	}
	delete[] events;

	//销毁流
	for (int i = 0; i < unitSize; i++) {
		hipStreamDestroy(streams[i]);
	}
	delete[] streams;

	//释放多余的内存
	hipFree(devicePartPtr);
	hipFree(deviceUnitPtr);

	//将合成结果复制
	if (auto result = hipMemcpy(buffer, hostBufferPtr, bufferSize * sizeof(float), hipMemcpyHostToHost)) {
		hipHostFree(hostBufferPtr);
		hipDeviceReset();
		return result;
	}

	//释放内存
	hipHostFree(hostBufferPtr);

	//重置设备
	hipDeviceReset();
	
	//返回成功
	return hipError_t::hipSuccess;
}

__global__ void computeUnit(
	float* unitMemBase, const float* unitBase, int unitLength, int totalThread)
{
	//运算系数
	constexpr float r = 0.5f;

	//定位GPU线程
	int index = totalThread * blockIdx.x + threadIdx.x;

	//定位处理目标
	if (index >= unitLength || index < 0) { return; }
	int nextIndex = index - 1;
	if (index == 0) { nextIndex = unitLength - 1; }

	//平均
	unitMemBase[index] = unitBase[index] * r + unitBase[nextIndex] * (1.f - r);
}

__global__ void attenuateUnit(
	float* unitMemBase, int unitLength, float sCoefficient, float eCoefficient, int totalThread)
{
	//定位GPU线程
	int index = totalThread * blockIdx.x + threadIdx.x;

	//定位处理目标
	if (index >= unitLength || index < 0) { return; }

	//计算衰减系数
	float coefficient = sCoefficient + (eCoefficient - sCoefficient) * (index / (float)(unitLength - 1));

	//衰减
	unitMemBase[index] *= coefficient;
}

#endif