#include "hip/hip_runtime.h"
﻿#include "KarPlusStrong.cuh"
#include <>

#if CUDA_ENABLED

__host__ hipError_t doSynthesis(
	const float* part, int partSize,
	float* buffer, int bufferSize,
	const int* unitArray, int unitSize)
{
	//无需计算
	if (partSize <= 1) { return hipError_t::hipErrorUnknown; }
	if (bufferSize <= 1) { return hipError_t::hipErrorUnknown; }
	if (unitSize == 0) { return hipError_t::hipErrorUnknown; }

	//选择设备
	int device = -1;
	hipDeviceProp_t deviceProp = cudaDevicePropDontCare;
	if (auto result = hipChooseDevice(&device, &deviceProp)) {
		return result;
	}
	if (auto result = hipSetDevice(device)) {
		return result;
	}

	//在主机上申请合成结果储存空间
	float* hostBufferPtr = nullptr;
	if (auto result = hipHostMalloc(&hostBufferPtr, bufferSize * sizeof(float), hipMemAttachGlobal)) {
		hipDeviceReset();
		return result;
	}

	//在设备上申请原始合成单元储存空间
	float* devicePartPtr = nullptr;
	if (auto result = hipMalloc(&devicePartPtr, partSize * sizeof(float))) {
		hipHostFree(hostBufferPtr);
		hipDeviceReset();
		return result;
	}

	//在设备上申请合成单元滤波后序列的储存空间
	float* deviceUnitPtr = nullptr;
	if (auto result = hipMalloc(&deviceUnitPtr, (size_t)unitSize * partSize * sizeof(float))) {
		hipHostFree(hostBufferPtr);
		hipFree(devicePartPtr);
		hipDeviceReset();
		return result;
	}

	//将原始合成单元存入设备
	if (auto result = hipMemcpy(devicePartPtr, part, partSize * sizeof(float), hipMemcpyHostToDevice)) {
		hipHostFree(hostBufferPtr);
		hipFree(devicePartPtr);
		hipFree(deviceUnitPtr);
		hipDeviceReset();
		return result;
	}

	//建立流
	hipStream_t* streams = nullptr;
	if (auto result = hipMalloc(&streams, unitSize * sizeof(hipStream_t))) {
		hipHostFree(hostBufferPtr);
		hipFree(devicePartPtr);
		hipFree(deviceUnitPtr);
		hipDeviceReset();
		return result;
	}
	for (int i = 0; i < unitSize; i++) {
		hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
	}

	//建立同步事件
	hipEvent_t* events = nullptr;
	if (auto result = hipMalloc(&events, unitSize * sizeof(hipEvent_t))) {
		//销毁流
		for (int i = 0; i < unitSize; i++) {
			hipStreamDestroy(streams[i]);
		}
		hipHostFree(hostBufferPtr);
		hipFree(devicePartPtr);
		hipFree(deviceUnitPtr);
		hipFree(streams);
		hipDeviceReset();
		return result;
	}
	for (int i = 0; i < unitSize; i++) {
		hipEventCreateWithFlags(&events[i], hipEventDisableTiming);
	}

	//获取设备属性
	auto& threadSizeInBlock = deviceProp.maxThreadsDim;
	int threadNumInABlock = threadSizeInBlock[0] * threadSizeInBlock[1] * threadSizeInBlock[2];

	//计算循环轮数
	int roundSize = (partSize / threadNumInABlock) + 1;

	//规划算子
	dim3 block(threadSizeInBlock[0], threadSizeInBlock[1], threadSizeInBlock[2]);

	//深度优先建立流计算任务
	int outDeviation = 0;
	for (int i = 0; i < unitSize; i++) {
		//在开始任务一前确认上一流已到达事件同步点
		if (i > 0) {
			hipStreamWaitEvent(streams[i], events[i - 1], cudaEventWaitExternal);
		}

		//任务一：对拼接单元依次低通滤波
		if (i == 0) {
			computeUnit <<<roundSize, block, 0, streams[i]>>> (
				&deviceUnitPtr[i * partSize],
				&devicePartPtr[0], partSize);
		}
		else {
			computeUnit <<<roundSize, block, 0, streams[i]>>> (
				&deviceUnitPtr[i * partSize],
				&deviceUnitPtr[(i - 1) * partSize], partSize);
		}

		//设置事件同步点
		hipEventRecordWithFlags(events[i], streams[i], hipEventRecordExternal);

		//在开始任务二前确认下一流已到达事件同步点
		if (i < unitSize - 1) {
			hipStreamWaitEvent(streams[i], events[i + 1], cudaEventWaitExternal);
		}

		//计算拼接单元长度
		int currentUnitLength = unitArray[i];
		if (currentUnitLength > partSize) { currentUnitLength = partSize; }

		//任务二：线性衰减
		int partRoundSize = (currentUnitLength / threadNumInABlock) + 1;
		attenuateUnit <<<partRoundSize, block, 0, streams[i]>>> (
			&deviceUnitPtr[i * partSize], currentUnitLength,
			1.f - outDeviation / (float)(bufferSize - 1),
			1.f - (outDeviation + currentUnitLength - 1) / (float)(bufferSize - 1));

		//任务三：将拼接单元拷贝到指定位置
		hipMemcpyAsync(
			&hostBufferPtr[outDeviation], &deviceUnitPtr[i * partSize], currentUnitLength, hipMemcpyDeviceToHost, streams[i]);
		outDeviation += unitArray[i];
	}

	//等待同步
	hipDeviceSynchronize();

	//销毁事件
	for (int i = 0; i < unitSize; i++) {
		hipEventDestroy(events[i]);
	}
	hipFree(events);

	//销毁流
	for (int i = 0; i < unitSize; i++) {
		hipStreamDestroy(streams[i]);
	}
	hipFree(streams);

	//释放多余的内存
	hipFree(devicePartPtr);
	hipFree(deviceUnitPtr);

	//将合成结果复制
	if (auto result = hipMemcpy(buffer, hostBufferPtr, bufferSize * sizeof(float), hipMemcpyHostToHost)) {
		hipHostFree(hostBufferPtr);
		hipDeviceReset();
		return result;
	}

	//释放内存
	hipHostFree(hostBufferPtr);

	//重置设备
	hipDeviceReset();
	
	//返回成功
	return hipError_t::hipSuccess;
}

__global__ void computeUnit(
	float* unitMemBase, const float* unitBase, int unitLength)
{
	//定位GPU线程
	int totalThread = blockDim.x * blockDim.y * blockDim.z;
	int threadIndex = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * blockDim.x + threadIdx.x;
	int index = totalThread * blockIdx.x + threadIndex;

	//定位处理目标
	if (index >= unitLength || index < 0) { return; }
	int nextIndex = index + 1;
	if (index == unitLength - 1) { nextIndex = 0; }

	//平均
	unitMemBase[index] = unitBase[index] + (unitBase[nextIndex] - unitBase[index]) / 2;
}

__global__ void attenuateUnit(
	float* unitMemBase, int unitLength, float sCoefficient, float eCoefficient)
{
	//定位GPU线程
	int totalThread = blockDim.x * blockDim.y * blockDim.z;
	int threadIndex = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * blockDim.x + threadIdx.x;
	int index = totalThread * blockIdx.x + threadIndex;

	//定位处理目标
	if (index >= unitLength || index < 0) { return; }

	//计算衰减系数
	float coefficient = sCoefficient + (eCoefficient - sCoefficient) * (index / (float)(unitLength - 1));

	//衰减
	unitMemBase[index] *= coefficient;
}

#endif