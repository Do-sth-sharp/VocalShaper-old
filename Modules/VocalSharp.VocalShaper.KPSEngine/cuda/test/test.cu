#include "hip/hip_runtime.h"
﻿#include "../KarPlusStrong.cuh"
#include "../../algorithm/KarPlusStrong.h"
#include <iostream>
#include <fstream>
#include <cstdio>
#include <vector>
#include <time.h>
#include <numeric>

inline int16_t PCMFloat32ToInt16FmtConvert(float sample)
{
	if (sample < -0.999999f) {
		return INT16_MIN;
	}
	else if (sample > 0.999999f) {
		return INT16_MAX;
	}
	else {
		return static_cast<int16_t>(sample * 32767.0f);
	}
}

inline float PCMInt16ToFloat32FmtConvert(int16_t sample)
{
	return static_cast<float>(sample) / -static_cast<float>(INT16_MIN);
}

int main()
{
	std::cout << "选择要加载的合成单元数据：";
	std::flush(std::cout);

	std::string unitPath;
	std::cin >> unitPath;

	FILE* unitFile = fopen(unitPath.c_str(), "r");
	if (!unitFile) {
		std::cout << "未能打开文件：" << unitPath << std::endl;
		system("pause");
		return -1;
	}

	fseek(unitFile, 0, SEEK_END);
	auto unitSize = ftell(unitFile);
	fseek(unitFile, 0, SEEK_SET);

	std::ifstream unitInputStream(unitFile);
	if (!unitInputStream.good()) {
		std::cout << "未能建立文件流：" << unitPath << std::endl;
		fclose(unitFile);
		system("pause");
		return -1;
	}

	if (unitSize > 0) {
		std::cout << "单元文件大小：" << unitSize << std::endl;
	}
	else {
		std::cout << "单元文件大小为零！" << std::endl;
		fclose(unitFile);
		system("pause");
		return -1;
	}

	int unitNum = unitSize / sizeof(int16_t);
	std::cout << "单元采样数：" << unitNum << std::endl;

	std::vector<float> unitList;
	unitList.resize(unitNum);

	for (int i = 0; i < unitNum; i++) {
		int16_t unit = 0;
		unitInputStream.read((char*)&unit, sizeof(int16_t));
		unitList[i] = PCMInt16ToFloat32FmtConvert(unit);
	}

	fclose(unitFile);

	float unitMean= std::accumulate(unitList.begin(), unitList.end(), 0.f) / unitList.size();
	for (int i = 0; i < unitNum; i++) {
		unitList[i] -= unitMean;
	}
	std::cout << "已加载合成单元！" << std::endl;

	int sampleRate = 48000;
	std::cout << "采样率：" << sampleRate << std::endl;

	std::cout << "输入合成频率：";
	std::flush(std::cout);
	float fre = 440.00;
	std::cin >> fre;

	std::cout << "输入合成时长：";
	std::flush(std::cout);
	float time = 10.0;
	std::cin >> time;

	int sampleSize = time * sampleRate;
	int partSize = 1 / fre * sampleRate;
	int partNum = std::ceil(sampleSize / (double)partSize);
	int lastPartSize = sampleSize % partSize;

	std::cout << "采样数：" << sampleSize << std::endl;
	std::cout << "片段数：" << partNum << std::endl;
	std::cout << "单个片段采样数：" << partSize << std::endl;
	std::cout << "末尾片段采样数：" << lastPartSize << std::endl;

	std::vector<int> partList;
	partList.resize(partNum);
	for (int i = 0; i < partNum; i++) {
		partList[i] = partSize;
	}
	partList[partNum - 1] = lastPartSize;

	std::vector<float> gpuBuffer;
	gpuBuffer.resize(sampleSize);
	std::vector<float> cpuBuffer;
	cpuBuffer.resize(sampleSize);

#if CUDA_ENABLED
	std::cout << "开始GPU合成！" << std::endl;
	clock_t gpuStartTime = clock();

	if (auto result = doSynthesisGPU(unitList.data(), unitSize, gpuBuffer.data(), sampleSize, partList.data(), partNum)) {
		std::cout << "GPU合成失败, 返回：" << (int)result << std::endl;
		system("pause");
		return -1;
	}

	clock_t gpuEndTime = clock();
	std::cout << "GPU合成结束, 用时：" << (gpuEndTime - gpuStartTime) / (float)CLOCKS_PER_SEC << std::endl;
#else
	std::cout << "不支持GPU合成！" << std::endl;
#endif // CUDA_ENABLED

	std::cout << "开始CPU合成！" << std::endl;
	clock_t cpuStartTime = clock();

	if (auto result = doSynthesisCPU(unitList.data(), unitSize, cpuBuffer.data(), sampleSize, partList.data(), partNum)) {
		std::cout << "CPU合成失败, 返回：" << (int)result << std::endl;
		system("pause");
		return -1;
	}

	clock_t cpuEndTime = clock();
	std::cout << "CPU合成结束, 用时：" << (cpuEndTime - cpuStartTime) / (float)CLOCKS_PER_SEC << std::endl;

	for (int i = sampleSize - lastPartSize; i < sampleSize; i++) {
		if (cpuBuffer[i] > 0.1) {
			std::cout << i << ":" << cpuBuffer[i] << std::endl;
			system("pause");
		}
	}

	std::cout << "输入GPU合成结果储存位置：";
	std::flush(std::cout);

	std::string gpuOutPath;
	std::cin >> gpuOutPath;

	FILE* gpuOutFile = fopen(gpuOutPath.c_str(), "w");
	if (!gpuOutFile) {
		std::cout << "未能打开文件：" << gpuOutPath << std::endl;
		system("pause");
		return -1;
	}

	std::ofstream gpuOutStream(gpuOutFile);
	if (!gpuOutStream.good()) {
		std::cout << "未能建立文件流：" << gpuOutPath << std::endl;
		fclose(gpuOutFile);
		system("pause");
		return -1;
	}

	for (int i = 0; i < sampleSize; i++) {
		int16_t temp = PCMFloat32ToInt16FmtConvert(gpuBuffer[i]);
		gpuOutStream.write((const char*)&temp, sizeof(int16_t));
	}

	fclose(gpuOutFile);
	std::cout << "已保存GPU合成结果！" << std::endl;

	std::cout << "输入CPU合成结果储存位置：";
	std::flush(std::cout);

	std::string cpuOutPath;
	std::cin >> cpuOutPath;

	FILE* cpuOutFile = fopen(cpuOutPath.c_str(), "w");
	if (!cpuOutFile) {
		std::cout << "未能打开文件：" << cpuOutPath << std::endl;
		system("pause");
		return -1;
	}

	std::ofstream cpuOutStream(cpuOutFile);
	if (!cpuOutStream.good()) {
		std::cout << "未能建立文件流：" << cpuOutPath << std::endl;
		fclose(cpuOutFile);
		system("pause");
		return -1;
	}

	for (int i = 0; i < sampleSize; i++) {
		int16_t temp = PCMFloat32ToInt16FmtConvert(cpuBuffer[i]);
		cpuOutStream.write((const char*)&temp, sizeof(int16_t));
	}

	fclose(cpuOutFile);
	std::cout << "已保存CPU合成结果！" << std::endl;

	system("pause");
	return 0;
}
